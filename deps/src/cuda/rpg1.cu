
#include <hip/hip_runtime.h>
#include <stddef.h>
//#include <stdio.h>
//#include <cuda.h>

#define NUM_THREADS 512
// cuda (gpu) reduce_pixels! for MeasuredSymmetric source and sink
__global__ void rpg1ker(double *sr1, long long *srCln, long long *Nsr, long long *Ncol,
                        long long *idx, long long *ttl,double *sn1, long long *snCln,
                        long long *snTar, long long *Ntar)
{
  long long i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < *Ntar)
  {
    long long j,k,l,m,mytarget=snTar[i]; // C-indexed target sink pixel
    for (j=0; j < *Nsr; j++) // loop through all source pixel indicies
    {
      if (idx[j]==mytarget) // if the source pixel index matches our target
      {
        for (k=0; k < *Ncol; k++) //loop over the columns for that pixel and
        {
          l=snCln[k]+mytarget; // the index of the target's kth column
          m=srCln[k]+j;        // the index of the source's kth column
          sn1[l]+=sr1[m]; // add to the target sink1 pixel columns
          ttl[l]+=1LL; // add one to the number of contributing source pixels
        }
      }
    }
  }
}

extern "C" void rpg1(double *sr1, long long *srCln, long long Nsr, long long Ncol,
          long long *idx, long long *ttl, double *sn1, long long *snCln, long long Nsn,
          long long *snTar, long long Ntar)
{
  double *d_sr1, *d_sn1;
  long long *d_srCln, *d_idx, *d_ttl, *d_snCln, *d_snTar;
  long long *d_Nsr, *d_Ncol, *d_Ntar;

  int sized = sizeof(double);
  int sizel = sizeof(long long);
  int Nthreads = NUM_THREADS;
  // determine how many blocks are necessary to visit all Ntar target
  // indexes using Nthreads threads.
  int Nblocks = (Ntar%Nthreads > 0) ? 1+Ntar/Nthreads : Ntar/Nthreads;

  hipMalloc( (void**)&d_sr1, Nsr*Ncol*sized);
  hipMalloc( (void**)&d_sn1, Nsn*Ncol*sized);
  hipMalloc( (void**)&d_srCln, Ncol*sizel);
  hipMalloc( (void**)&d_idx, Nsr*sizel);
  hipMalloc( (void**)&d_ttl, Nsn*Ncol*sizel);
  hipMalloc( (void**)&d_snCln, Ncol*sizel);
  hipMalloc( (void**)&d_snTar, Ntar*sizel);
  hipMalloc( (void**)&d_Nsr,  sizel);
  hipMalloc( (void**)&d_Ncol, sizel);
  hipMalloc( (void**)&d_Ntar, sizel);


  hipMemcpy( d_sr1,   sr1,   Nsr*Ncol*sized, hipMemcpyHostToDevice);
  hipMemcpy( d_sn1,   sn1,   Nsn*Ncol*sized, hipMemcpyHostToDevice);
  hipMemcpy( d_srCln, srCln, Ncol*sizel,     hipMemcpyHostToDevice);
  hipMemcpy( d_idx,   idx,   Nsr*sizel,      hipMemcpyHostToDevice);
  hipMemcpy( d_ttl,   ttl,   Nsn*Ncol*sizel, hipMemcpyHostToDevice);
  hipMemcpy( d_snCln, snCln, Ncol*sizel,     hipMemcpyHostToDevice);
  hipMemcpy( d_snTar, snTar, Ntar*sizel,     hipMemcpyHostToDevice);
  hipMemcpy( d_Nsr,   &Nsr,   sizel,          hipMemcpyHostToDevice);
  hipMemcpy( d_Ncol,  &Ncol,  sizel,          hipMemcpyHostToDevice);
  hipMemcpy( d_Ntar,  &Ntar,  sizel,          hipMemcpyHostToDevice);

  rpg1ker<<<Nblocks,Nthreads>>>(d_sr1,d_srCln,d_Nsr,d_Ncol,d_idx,d_ttl,d_sn1,d_snCln,d_snTar,d_Ntar);

  hipDeviceSynchronize();

  hipMemcpy( sn1, d_sn1, Nsn*Ncol*sized, hipMemcpyDeviceToHost);
  hipMemcpy( ttl, d_ttl, Nsn*Ncol*sized, hipMemcpyDeviceToHost);

  hipFree( d_Ntar );
  hipFree( d_Ncol );
  hipFree( d_Nsr  );
  hipFree( d_snTar);
  hipFree( d_snCln);
  hipFree( d_ttl  );
  hipFree( d_idx  );
  hipFree( d_srCln);
  hipFree( d_sn1  );
  hipFree( d_sr1  );
}
