
#include <hip/hip_runtime.h>
#include <stddef.h>
//#include <stdio.h>
//#include <cuda.h>

#define NUM_THREADS 512
// cuda (gpu) reduce_pixels! for MeasuredSymmetric source and sink
__global__ void rpg2ker(double *sr1, double *sr2, long long *srCln, long long *Nsr, long long *Ncol,
                        long long *idx, long long *ttl,double *sn1, double *sn2, long long *snCln,
                        long long *snTar, long long *Ntar)
{
  long long i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < *Ntar)
  {
    long long j,k,l,m,mytarget=snTar[i]; // C-indexed target sink pixel
    for (j=0; j < *Nsr; j++) // loop through all source pixel indicies
    {
      if (idx[j]==mytarget) // if the source pixel index matches our target
      {
        for (k=0; k < *Ncol; k++) //loop over the columns for that pixel and
        {
          l=snCln[k]+mytarget; // the index of the target's kth column
          m=srCln[k]+j;        // the index of the source's kth column
          sn1[l]+=sr1[m]; // add to the target sink1 pixel columns
          sn2[l]+=sr2[m]; // add to the target sink2 pixel columns
          ttl[l]+=1LL; // add one to the number of contributing source pixels
        }
      }
    }
  }
}

extern "C" void rpg2(double *sr1, double *sr2, long long *srCln, long long Nsr, long long Ncol,
          long long *idx, long long *ttl, double *sn1, double *sn2, long long *snCln, long long Nsn,
          long long *snTar, long long Ntar)
{
  double *d_sr1, *d_sr2, *d_sn1, *d_sn2;
  long long *d_srCln, *d_idx, *d_ttl, *d_snCln, *d_snTar;
  long long *d_Nsr, *d_Ncol, *d_Ntar;

  int sized = sizeof(double);
  int sizel = sizeof(long long);
  int Nthreads = NUM_THREADS;
  // determine how many blocks are necessary to visit all Ntar target
  // indexes using Nthreads threads.
  int Nblocks = (Ntar%Nthreads > 0) ? 1+Ntar/Nthreads : Ntar/Nthreads;

  //int gpuCount, gpu;
  //CUresult res;
  //CUdevice dev;
  //CUcontext ctx;
  //size_t dev_free, dev_total;
  
  //cuInit(0);
  //cuDeviceGetCount(&gpuCount);
  //printf( (gpuCount>1)?"There are %d GPUs present.\n":"There is %d GPU present.\n" ,gpuCount);
  //for (gpu=0; gpu<gpuCount; gpu++){
  //  cuDeviceGet(&dev,gpu);
  //  cuCtxCreate(&ctx,0,dev);
  //  res=cuMemGetInfo(&dev_free,&dev_total);
  //  if (res != CUDA_SUCCESS)
  //    printf("  cuMemGetInfo failed for GPU %d! (status=%x)\n",gpu+1,res);
  //  printf(" GPU %d has %lu bytes free of %lu total bytes.\n",gpu+1,dev_free,dev_total);
  //  cuCtxDetach(ctx);
  //}

 // printf("starting allocation of %d bytes on device", 2*Nsr*Ncol*sized+2*Nsn*Ncol*sized+2*Ncol*sizel+Nsn*Ncol*sizel+Nsr*sizel+Ntar*sizel+3*sizel);

  hipMalloc( (void**)&d_sr1, Nsr*Ncol*sized);
  hipMalloc( (void**)&d_sr2, Nsr*Ncol*sized);
  hipMalloc( (void**)&d_sn1, Nsn*Ncol*sized);
  hipMalloc( (void**)&d_sn2, Nsn*Ncol*sized);
  hipMalloc( (void**)&d_srCln, Ncol*sizel);
  hipMalloc( (void**)&d_idx, Nsr*sizel);
  hipMalloc( (void**)&d_ttl, Nsn*Ncol*sizel);
  hipMalloc( (void**)&d_snCln, Ncol*sizel);
  hipMalloc( (void**)&d_snTar, Ntar*sizel);
  hipMalloc( (void**)&d_Nsr,  sizel);
  hipMalloc( (void**)&d_Ncol, sizel);
  hipMalloc( (void**)&d_Ntar, sizel);


  hipMemcpy( d_sr1,   sr1,   Nsr*Ncol*sized, hipMemcpyHostToDevice);
  hipMemcpy( d_sr2,   sr2,   Nsr*Ncol*sized, hipMemcpyHostToDevice);
  hipMemcpy( d_sn1,   sn1,   Nsn*Ncol*sized, hipMemcpyHostToDevice);
  hipMemcpy( d_sn2,   sn2,   Nsn*Ncol*sized, hipMemcpyHostToDevice);
  hipMemcpy( d_srCln, srCln, Ncol*sizel,     hipMemcpyHostToDevice);
  hipMemcpy( d_idx,   idx,   Nsr*sizel,      hipMemcpyHostToDevice);
  hipMemcpy( d_ttl,   ttl,   Nsn*Ncol*sizel, hipMemcpyHostToDevice);
  hipMemcpy( d_snCln, snCln, Ncol*sizel,     hipMemcpyHostToDevice);
  hipMemcpy( d_snTar, snTar, Ntar*sizel,     hipMemcpyHostToDevice);
  hipMemcpy( d_Nsr,   &Nsr,   sizel,          hipMemcpyHostToDevice);
  hipMemcpy( d_Ncol,  &Ncol,  sizel,          hipMemcpyHostToDevice);
  hipMemcpy( d_Ntar,  &Ntar,  sizel,          hipMemcpyHostToDevice);

  rpg2ker<<<Nblocks,Nthreads>>>(d_sr1,d_sr2,d_srCln,d_Nsr,d_Ncol,d_idx,d_ttl,d_sn1,d_sn2,d_snCln,d_snTar,d_Ntar);

  hipDeviceSynchronize();

  hipMemcpy( sn1, d_sn1, Nsn*Ncol*sized, hipMemcpyDeviceToHost);
  hipMemcpy( sn2, d_sn2, Nsn*Ncol*sized, hipMemcpyDeviceToHost);
  hipMemcpy( ttl, d_ttl, Nsn*Ncol*sized, hipMemcpyDeviceToHost);

  hipFree( d_Ntar );
  hipFree( d_Ncol );
  hipFree( d_Nsr  );
  hipFree( d_snTar);
  hipFree( d_snCln);
  hipFree( d_ttl  );
  hipFree( d_idx  );
  hipFree( d_srCln);
  hipFree( d_sn2  );
  hipFree( d_sn1  );
  hipFree( d_sr2  );
  hipFree( d_sr1  );
}
